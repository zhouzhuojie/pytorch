#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/ExpandUtils.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseCsrTensorImpl.h>
#include <ATen/SparseCsrTensorUtils.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/WrapDimUtilsMulti.h>
#include <ATen/native/BinaryOps.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <type_traits>

#include <THC/THCTensorMathPointwise.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.cuh>
#include <c10/cuda/CUDACachingAllocator.h>

#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>

namespace at {
namespace native {

using namespace at::sparse_csr;
// certain utiliy functions are usable from sparse COO.
using namespace at::sparse;

// // Functions for matrix multiplication.

// Tensor addmm_sparse_csr_dense_cuda(
//     const Tensor& self,
//     const SparseCsrTensor& sparse,
//     const Tensor& dense,
//     const Scalar& beta,
//     const Scalar& alpha) {
//   Tensor r = at::empty({0}, self.options());
//   at::addmm_out(r, self, sparse, dense, beta, alpha);
//   return r;
// }

//@aocsa TODO: refactor common cpu/cuda source code
//@aocsa TODO 1: checks and commong pre_processing like asinh!

Tensor& add_out_dense_sparse_csr_cuda(
    Tensor& output,
    const Tensor& dense,
    const SparseCsrTensor& src,
    const Scalar& alpha) {
  AT_ASSERT(dense.layout() == kStrided);
  AT_ASSERT(src.is_sparse_csr());
  AT_ASSERT(dense.is_cuda());

  TORCH_CHECK(
      output.is_contiguous(),
      "out argument must be contiguous, but got: ",
      output.suggest_memory_format());
  TORCH_CHECK(
      output.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      output.device());

  TORCH_CHECK(
      src.is_cuda(),
      "add: expected 'other' to be a CUDA tensor, but got tensor on device: ",
      src.device());

  TORCH_CHECK(
      dense.sizes().equals(src.sizes()),
      "add: expected 'self' and 'other' to have same size, but self has size ",
      dense.sizes(),
      " while other has size ",
      src.sizes(),
      " (FYI: op2-sparse addition does not currently support broadcasting)");

  auto commonDtype = promoteTypes(dense.scalar_type(), src.scalar_type());
  TORCH_CHECK(
      canCast(commonDtype, output.scalar_type()),
      "Can't convert result type ",
      commonDtype,
      " to output ",
      output.scalar_type(),
      " in add operation");

  Tensor src_values = src.values().to(commonDtype);
  Tensor src_crow_indices = src.crow_indices();
  Tensor src_col_indices = src.col_indices();

  output.resize_as_(dense);
  Tensor resultBuffer = output;
  Tensor valuesBuffer = src_values.to(commonDtype);

  if (output.scalar_type() != commonDtype) {
    resultBuffer = dense.to(commonDtype);
  } else if (!is_same_tensor(output, dense)) {
    resultBuffer.copy_(dense);
  }
  AT_DISPATCH_ALL_TYPES(
      commonDtype,
      "add_out_op2_sparse_csr",
      [&src_values, &output, &alpha, &src_crow_indices, &src_col_indices]() {
        AT_DISPATCH_INDEX_TYPES(
            src_crow_indices.scalar_type(),
            "csr_add_out_crow_indices",
              [&src_values, &output, &alpha, &src_crow_indices, &src_col_indices]() {
                scalar_t* values_accessor = src_values.data_ptr<scalar_t>();
                scalar_t* out_ptr = output.data_ptr<scalar_t>();
                scalar_t cast_value = alpha.to<scalar_t>();

                index_t* crow_indices_accessor = src_crow_indices.data_ptr<index_t>();
                index_t* col_indices_accessor = src_col_indices.data_ptr<index_t>();
                int64_t out_storage_offset = output.storage_offset();

                auto out_strides = output.strides();
                int64_t out_strides0 = out_strides[0];
                int64_t out_strides1 = out_strides[1];

                hipStream_t stream = at::cuda::getCurrentCUDAStream();
                auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
                auto policy = thrust::cuda::par(allocator).on(stream);

               thrust::for_each(
                    policy,
                    thrust::make_counting_iterator(int64_t(0)),
                    thrust::make_counting_iterator(int64_t(src_crow_indices.size(0) - 1)),
                    [values_accessor,
                    crow_indices_accessor,
                    col_indices_accessor,
                    out_ptr,
                    out_storage_offset,
                    out_strides0,
                    cast_value,
                    out_strides1
                    ]__device__(int64_t irow) {
                        int32_t start_index = crow_indices_accessor[irow];
                        int32_t end_index = crow_indices_accessor[irow + 1];

                        for (int i = start_index; i < end_index; ++i) {
                            auto icol = col_indices_accessor[i];
                            auto index = out_storage_offset + irow * out_strides0 + icol * out_strides1;
                            out_ptr[index] += cast_value * values_accessor[i];
                        }
                    });
              });
      });
  return output;
}

Tensor& add_out_sparse_csr_cuda(
    const Tensor& self,
    const SparseCsrTensor& other,
    const Scalar& alpha,
    SparseCsrTensor& out) {
  if (self.layout() == kStrided) {
    return add_out_dense_sparse_csr_cuda(out, self, other, alpha);
  } else {
    TORCH_CHECK(
        false,
        "NotImplementedError: Addition of sparse CSR tensors is not yet implemented.")
  }
  return out;
}

} // namespace native
} // namespace at
